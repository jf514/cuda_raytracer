#include "hip/hip_runtime.h"
#include "camera.h"
#include "common.h"
#include "hit.h"
#include "parallel.h"
#include "pcg.h"
#include "ray.h"
#include "sphere.h"
#include "timer.h"
#include "utils.h"
#include "world.h"

#include <cstdio>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <fstream>
#include <iostream>
#include <thread>

#define CHECK_CUDA_ERRORS(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        std::cerr << hipGetErrorString(result) << "\n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__HD__ Vector3 Color(const Ray& r, World* world, void* rand_state) {
    Ray cur_ray = r;
    Vector3 cur_attenuation = Vector3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        Hit hit = collideWorld(cur_ray, 0.001f, max_flt, world);
        if (hit.hit) {
            Ray scattered;
            Vector3 attenuation;
            if(hit.material->scatter(cur_ray, hit, attenuation, scattered, rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return Vector3(0.0,0.0,0.0);
            }
        }
        else {
            float t = 0.5f*(cur_ray.dir.y + 1.0f);
            Vector3 c = (1.0f-t)*Vector3(1.0, 1.0, 1.0) + t*Vector3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return Vector3(0.0,0.0,0.0); // exceeded recursion
}

__host__ __device__ Vector3 RenderImpl(Vector3* img, Camera cam, 
        Sphere* scene, int num_spheres, Ray ray){
    for(int i = 0; i < num_spheres; ++i){
        Hit h = collide(ray, scene[i], 0, max_flt);
        if (h.hit) {
            //printdb("hit pos", h.position);
            return 0.5f*Vector3(h.n.x+1.0f, h.n.y+1.0f, h.n.z+1.0f);
        }
    }

    float t = 0.5f*(ray.dir.y + 1.0f);
    return (1.0f-t)*Vector3(1.0, 1.0, 1.0) + t*Vector3(0.5, 0.7, 1.0);
}

__global__ void RenderInit(int max_x, int max_y, World** world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if((i == 0) && (j == 0)){
        *world = CreateWorld();
    }

    if((i >= max_x) || (j >= max_y)) 
        return;
    
    int pixel_index = j*max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void RenderCleanup(World** world){
    delete *world;
}

// GPU version of the same function
__global__ void Render(Vector3* img, Camera cam, World** world_d, 
                        hiprandState* rand_state){
    int tx = blockIdx.x*blockDim.x+threadIdx.x;
    int ty = blockIdx.y*blockDim.y+threadIdx.y;

    if((tx >= cam.image_width) || (ty >= cam.get_height())){ 
        return;
    }

    Vector3 color(0,0,0);
    for(int s = 0; s < cam.samples_per_pixel; ++s){
        int pixel_index = ty*cam.image_width + tx;
        hiprandState* local_rand_state = &rand_state[pixel_index];
        float rnd_x = hiprand_uniform(local_rand_state);
        float rnd_y = hiprand_uniform(local_rand_state);
        Ray ray = cam.get_ray(tx, ty, rnd_x, rnd_y);
        color += Color(ray, *world_d, local_rand_state);
    }

    color = color/float(cam.samples_per_pixel);
    img[tx + cam.image_width*ty] = color;
}

void RenderCPU(Vector3* img, Camera cam, World* world){
    int h = cam.image_width;
    int w = cam.image_width;

    constexpr int tile_size = 16;
    int num_tiles_x = (w + tile_size - 1) / tile_size;
    int num_tiles_y = (h + tile_size - 1) / tile_size;

    //hiprandState cstate;
    //hiprand_init(1984, 0, 0, &cstate);

    parallel_for([&](const Vector2i &tile) {
        // Use a different rng stream for each thread.
        pcg32_state rng = init_pcg32(tile[1] * num_tiles_x + tile[0]);
        int x0 = tile[0] * tile_size;
        int x1 = min(x0 + tile_size, w);
        int y0 = tile[1] * tile_size;
        int y1 = min(y0 + tile_size, h);
        for (int y = y0; y < y1; y++) {
            for (int x = x0; x < x1; x++) {
                Vector3 color(0,0,0);
                for(int s = 0; s < cam.samples_per_pixel; ++s){
                    float rnd_x = next_pcg32_real<float>(rng);
                    float rnd_y = next_pcg32_real<float>(rng);
                    Ray ray = cam.get_ray(x, y, rnd_x, rnd_y);
                    color += Color(ray, world, &rng);
                }
                color /= cam.samples_per_pixel;
                img[x + w*y] = color;
            }
        }
    }, Vector2i(num_tiles_x, num_tiles_y));
}

// __global__ void WorldCreate(World** world){
//     *world = CreateWorld();
// }

int main(int argc, char* argv[]){

    // Image side length - for this image size 
    // we expect CPU to be faster. For my architecture
    // I don't see the GPU going faster until 
    // N = 8 * 512, if we include memory transfer. However,
    // excluding transfer we see a speed up of a factor of
    // 100.
    const int N = 512;

    // Set up camera
    Camera cam;
    cam.image_width = N;
    cam.lookat = Vector3(0,0,1);
    cam.lookfrom = Vector3(0,0,0);

    cam.initialize();
       
    // Represent images as 1-D array of size N*N
    Vector3* img_h = new Vector3[N*N];

    // Render on GPU or CPU?
    float deltaT = 0;
    if(argc > 1 && std::string(argv[1]) == "-cpu"){
        std::cout << "Rendering on CPU...\n";

        World* w_h = CreateWorld();
 
        //int num_threads = std::thread::hardware_concurrency();
        int num_threads = 50;
        parallel_init(num_threads);

        Timer timer;
        tick(timer);
        RenderCPU(img_h, cam, w_h);
        deltaT = tick(timer);

        parallel_cleanup();
    } else {
        std::cout << "Rendering on GPU...\n";

        // Allocate device memory.
        Vector3 *img_d;
        CHECK_CUDA_ERRORS(hipMalloc(&img_d, N*N*sizeof(Vector3)));

        //Sphere* scene_d;
        //CHECK_CUDA_ERRORS(hipMalloc(&scene_d, num_spheres*sizeof(Sphere)));
        //CHECK_CUDA_ERRORS(hipMemcpy(scene_d,scene_h,num_spheres*sizeof(Sphere),hipMemcpyHostToDevice));

        // allocate random state
        hiprandState* rand_state_d;
        CHECK_CUDA_ERRORS(hipMalloc((void **)&rand_state_d, N*N*sizeof(hiprandState)));

        const dim3 threads(32, 32);
        const dim3 blocks(
            N/threads.x,
            N/threads.y
        );

        Timer timer;
        tick(timer);

        // *NOTE - this takes almost 5 ms, so see if we can reuse this
        // state when attempting RT
        World** world_d;
        CHECK_CUDA_ERRORS(hipMalloc(&world_d,sizeof(World**)));
        //WorldCreate<<<1,1>>>(world_d);
        RenderInit<<<blocks, threads>>>(N, N, world_d, rand_state_d);

        CHECK_CUDA_ERRORS(hipGetLastError());
        CHECK_CUDA_ERRORS(hipDeviceSynchronize());

        Render<<<blocks, threads>>>(img_d, cam, world_d, rand_state_d);
        CHECK_CUDA_ERRORS(hipGetLastError());
        CHECK_CUDA_ERRORS(hipDeviceSynchronize());
        deltaT = tick(timer);

        RenderCleanup<<<blocks, threads>>>(world_d);
        world_d = nullptr;

        // Copy data back to host.
        CHECK_CUDA_ERRORS(hipMemcpy(img_h, img_d, N*N*sizeof(Vector3), hipMemcpyDeviceToHost));

        // Free memory.
        CHECK_CUDA_ERRORS(hipFree(img_d));

        // Optional in a single threaded context, but use included for
        // demo purposes.
        hipDeviceReset();
    }

    std::cout << "Finished. Took: " << 1000.*deltaT << " milliseconds.\n";

    WritePPM("test_out.ppm", img_h, N);
    delete[] img_h;

    //delete that spheres

}