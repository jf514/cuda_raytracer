#include "hip/hip_runtime.h"
#include "common.h"

#include <cstdio>
#include <fstream>
#include <iostream>

__global__ void Render(Vector3* img, int N){
    int tx = blockIdx.x*blockDim.x+threadIdx.x;
    int ty = blockIdx.y*blockDim.y+threadIdx.y;

    const float x = tx - 0.5f*N;
    float y = ty - 0.5f*N;

    float rt = std::sqrt( x*x + y*y );
    if(rt < 0.25f * N) {
        img[tx + N*ty] = Vector3(0.5,0,0);
    } else {
        img[tx + N*ty] = Vector3(0.5,1,1);
    }
}

// CPU version of the same function
void RenderCPU(Vector3* img, int N){
    for(int tx = 0; tx < N; ++tx){
        for(int ty = 0; ty < N; ++ty){
            float x = tx - 0.5*N;
            float y = ty - 0.5*N;
        
            float rt = std::sqrt( x*x + y*y );
            if(rt < 0.25 * N) {
                img[tx + N*ty] = Vector3(0.5,0,0);
            } else {
                img[tx + N*ty] = Vector3(1,1,1);
            }
        }
    }
}

int main(int argc, char* argv[]){

    // Image side length - for this image size 
    // we expect CPU to be faster. For my architecture
    // I don't see the GPU going faster until 
    // N = 8 * 512, if we include memory transfer. However,
    // excluding transfer we see a speef up of a factor of
    // 100.
    const int N = 512;

    // Represent images as 1-D array of size N*N
    Vector3* img_h = new Vector3[N*N];

    // Render on GPU or CPU?
    float deltaT = 0;
    if(argc > 1 && std::string(argv[1]) == "-cpu"){
        std::cout << "Rendering on CPU...\n";
 
        Timer timer;
        tick(timer);
        RenderCPU(img_h, N);

        deltaT = tick(timer);

    } else {
        std::cout << "Rendering on GPU...\n";

        // Device
        Vector3 *img_d;
        hipMalloc(&img_d, N*N*sizeof(Vector3));

        const dim3 blockThreadDist(32, 32);
        const dim3 numBlocks(
            N/blockThreadDist.x,
            N/blockThreadDist.y
        );

        Timer timer;
        tick(timer);
        Render<<<numBlocks, blockThreadDist>>>(img_d, N);
        deltaT = tick(timer);

        hipMemcpy(img_h, img_d, N*N*sizeof(Vector3), hipMemcpyDeviceToHost);
        // Free memory
        hipFree(img_d);
    }

    std::cout << "Finished. Took: " << 1000.*deltaT << " milliseconds.\n";

    WritePPM("test_out.ppm", img_h, N);
    delete[] img_h;

    return 0;
}